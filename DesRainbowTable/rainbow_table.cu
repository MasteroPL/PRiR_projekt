#include "rainbow_table.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

/// <summary>
/// Alokuje pamiec dla tablicy teczowej o wybranym rozmiarze
/// </summary>
/// <param name="key_size">Rozmiar (w bajtach) kluczy (zawsze 8)</param>
/// <param name="encoded_password_size">Rozmiar (w bajtach) zaszyfrowanej tre�ci hasla</param>
/// <param name="num_of_entries">Liczba wpisow do zaalokowania</param>
/// <returns>Wskaznik na zaalokowan� pamiec</returns>
rainbow_table_t* RainbowTable_allocate(short key_size, short encoded_password_size, int num_of_entries) {
	rainbow_table_t* result = (rainbow_table_t*)malloc(sizeof(rainbow_table_t));
	if (result == NULL) {
		return NULL;
	}
	int per_chunk = num_of_entries;
	int entry_size = sizeof(unsigned char) * key_size + sizeof(unsigned char) * encoded_password_size;

	result->key_size = key_size;
	result->encoded_password_size = encoded_password_size;
	result->nodes_size = num_of_entries;

	result->nodes = (rainbow_table_node_t*)malloc(sizeof(rainbow_table_node_t) * num_of_entries);

	if (result->nodes == NULL) {
		free(result);
		return NULL;
	}

	int cur_malloc_size = entry_size * num_of_entries;

	// Trzeba podzielic rezerwowan� pami�c
	if (cur_malloc_size > MAX_MALLOC_SIZE) {
		int num_of_chunks = cur_malloc_size / MAX_MALLOC_SIZE;
		if (cur_malloc_size % MAX_MALLOC_SIZE > 0) {
			num_of_chunks++;
		}
		per_chunk = num_of_entries / num_of_chunks;
		if (num_of_entries % num_of_chunks > 0) {
			num_of_chunks++;
			per_chunk = num_of_entries / num_of_chunks;
			if (num_of_entries % num_of_chunks > 0) {
				per_chunk++;
			}
		}
		int tmp_entries = num_of_entries;

		result->_nodes_data_ref = (unsigned char**)malloc(sizeof(char*) * num_of_chunks);
		result->_num_of_refs = num_of_chunks;

		if (result->_nodes_data_ref == NULL) {
			free(result);
			return NULL;
		}

		result->_ref_sizes = (int*)malloc(sizeof(int) * num_of_chunks);

		if (result->_ref_sizes == NULL) {
			free(result->_nodes_data_ref);
			free(result);
			return NULL;
		}

		for (int i = 0; i < num_of_chunks; i++) {
			if (tmp_entries > per_chunk) {
				result->_nodes_data_ref[i] = (unsigned char*)malloc(
					entry_size * per_chunk
				);
				result->_ref_sizes[i] = per_chunk;
			}
			else {
				result->_nodes_data_ref[i] = (unsigned char*)malloc(
					entry_size * tmp_entries
				);
				result->_ref_sizes[i] = per_chunk;
			}

			if (result->_nodes_data_ref[i] == NULL) {
				for (int j = 0; j < i; j++) {
					free(result->_nodes_data_ref[i]);
				}
				free(result->_nodes_data_ref);
				free(result->_ref_sizes);
				free(result);
				return NULL;
			}

			tmp_entries -= per_chunk;
		}
	}
	// Wszystko mo�na zarezerwowa� w jednym miejscu
	else {
		result->_nodes_data_ref = (unsigned char**)malloc(sizeof(unsigned char*));
		if (result->_nodes_data_ref == NULL) {
			free(result);
			return NULL;
		}
		result->_ref_sizes = (int*)malloc(sizeof(int));
		if (result->_ref_sizes == NULL) {
			free(result->_nodes_data_ref);
			free(result);
			return NULL;
		}
		result->_nodes_data_ref[0] = (unsigned char*)malloc(cur_malloc_size);
		if (result->_nodes_data_ref[0] == NULL) {
			free(result->_nodes_data_ref);
			free(result->_ref_sizes);
			free(result);
			return NULL;
		}
		result->_num_of_refs = 1;
		result->_ref_sizes[0] = num_of_entries;
	}

	unsigned char** tmp_data_ref = result->_nodes_data_ref;

	result->nodes[0].key = &(tmp_data_ref[0][0]);
	result->nodes[0].encoded_password = &(tmp_data_ref[0][key_size]);

	// Przypisanie wskaza� na odpowiednie miejsca w pami�ci do wpisywania bajt�w kluczy i zaszyfrowanych hase�
	int chunk_index = 0;
	int tmp_i = 1;
	for (int i = 1; i < num_of_entries; i++, tmp_i++) {
		if (i % per_chunk == 0) {
			chunk_index++;
			tmp_i = 0;
		}

		result->nodes[i].key = (unsigned char*)&(tmp_data_ref[chunk_index][tmp_i * (key_size + encoded_password_size)]);
		result->nodes[i].encoded_password = (unsigned char*)&(tmp_data_ref[chunk_index][tmp_i * (key_size + encoded_password_size) + key_size]);
	}

	return result;
}

void RainbowTable_free(rainbow_table_t* self) {
	for (int i = 0; i < self->_num_of_refs; i++) {
		free(self->_nodes_data_ref[i]);
	}

	free(self->_ref_sizes);
	free(self->_nodes_data_ref);
	free(self->nodes);
}

int RainbowTable_write_to_file(rainbow_table_t* self, const char* filename) {
	FILE* f = fopen(filename, "wb");

	// Sprawdzenie czy plik jest otwarty
	if (f == NULL) {
		return 1;
	}

	// Wpisywanie meta danych
	char meta_data[8];
	int* num_of_entries = (int*)&(meta_data[0]);
	short* key_size = (short*)&(meta_data[4]);
	short* password_size = (short*)&(meta_data[6]);

	*num_of_entries = self->nodes_size;
	*key_size = self->key_size;
	*password_size = self->encoded_password_size;

	fwrite(meta_data, 1, 8, f);

	for (int i = 0; i < self->_num_of_refs; i++) {
		fwrite(self->_nodes_data_ref[i], sizeof(unsigned char) * self->key_size + sizeof(unsigned char) * self->encoded_password_size, self->_ref_sizes[i], f);
	}

	fclose(f);
	return 0;
}


rainbow_table_t* RainbowTable_read_from_file(const char* filename) {
	FILE* f = fopen(filename, "rb");

	// Czy plik otwarty
	if (f == NULL) {
		return NULL;
	}

	char meta_data[8];
	
	fread(meta_data, 1, 8, f);
	int* num_of_entries = (int*)&(meta_data[0]);
	short* key_size = (short*)&(meta_data[4]);
	short* password_size = (short*)&(meta_data[6]);

	rainbow_table_t* t = RainbowTable_allocate(*key_size, *password_size, *num_of_entries);

	if (t == NULL) {
		// Nie mo�na zarezerwowa� pami�ci
		fclose(f);
		return NULL;
	}

	rainbow_table_node_t tmp_node;
	for (int i = 0; i < *num_of_entries; i++) {
		tmp_node = t->nodes[i];

		// Pami�� zosta�a ju� zaalokowana
		
		if (fread(tmp_node.key, 1, t->key_size, f) < 0) {
			// Nieprawid�owy plik
			fclose(f);
			return NULL;
		}
		if (fread(tmp_node.encoded_password, 1, t->encoded_password_size, f) < 0) {
			// Nieprawid�owy plik
			fclose(f);
			return NULL;
		}
	}

	fclose(f);
	return t;
}


void RainbowTable_cuda_allocate(rainbow_table_t* ref_rainbow_table, unsigned char*** keys_pointers, unsigned char*** encoded_passwords_pointers, unsigned char*** origins_refs) {
	rainbow_table_t* ref = ref_rainbow_table;
	int entry_size = sizeof(unsigned char) * ref->key_size + sizeof(unsigned char) * ref->encoded_password_size;

	hipMalloc(keys_pointers, sizeof(unsigned char*) * ref->nodes_size);
	hipMalloc(encoded_passwords_pointers, sizeof(unsigned char*) * ref->nodes_size);
	hipMalloc(origins_refs, sizeof(unsigned char*) * ref->_num_of_refs);

	unsigned char** h_keys_pointers = (unsigned char**)malloc(sizeof(unsigned char*) * ref->nodes_size);
	unsigned char** h_encoded_passwords_pointers = (unsigned char**)malloc(sizeof(unsigned char*) * ref->nodes_size);
	unsigned char** h_origins_refs = (unsigned char**)malloc(sizeof(unsigned char*) * ref->_num_of_refs);
	for (int i = 0; i < ref->_num_of_refs; i++) {
		hipMalloc(&((h_origins_refs)[i]), entry_size * ref->_ref_sizes[i]);
	}
	hipMemcpy(*origins_refs, h_origins_refs, sizeof(unsigned char*) * ref->_num_of_refs, hipMemcpyHostToDevice);
	hipMemcpy(h_origins_refs, *origins_refs, sizeof(unsigned char*) * ref->_num_of_refs, hipMemcpyDeviceToHost);

	int cur_size = ref->_ref_sizes[0];
	int cur_tmp = 0;
	int cur_ref = 0;
	for (int i = 0; i < ref->nodes_size; i++, cur_tmp++) {
		if (cur_tmp == cur_size) {
			cur_tmp = 0;
			cur_ref++;
			cur_size = ref->_ref_sizes[cur_ref];
		}

		if (i == 1024) {
			int c = 0;
		}

		int a = cur_tmp * entry_size;
		int b = (cur_tmp * entry_size) + ref->key_size;
		(h_keys_pointers)[i] = &(h_origins_refs)[cur_ref][cur_tmp * entry_size];
		(h_encoded_passwords_pointers)[i] = &(h_origins_refs)[cur_ref][(cur_tmp * entry_size) + ref->key_size];
	}

	hipMemcpy(*keys_pointers, h_keys_pointers, sizeof(unsigned char**) * ref->nodes_size, hipMemcpyHostToDevice);
	hipMemcpy(*encoded_passwords_pointers, h_encoded_passwords_pointers, sizeof(unsigned char**) * ref->nodes_size, hipMemcpyHostToDevice);

	free(h_keys_pointers);
	free(h_encoded_passwords_pointers);
	free(h_origins_refs);
}

void RainbowTable_cuda_copy_results_to_host(rainbow_table_t* ref_rainbow_table, unsigned char** origins_refs) {
	rainbow_table_t* ref = ref_rainbow_table;
	int entry_size = sizeof(unsigned char) * ref->key_size + sizeof(unsigned char) * ref->encoded_password_size;

	unsigned char** h_origins_refs = (unsigned char** )malloc(sizeof(unsigned char*) * ref->_num_of_refs);
	hipMemcpy(h_origins_refs, origins_refs, sizeof(unsigned char*) * ref->_num_of_refs, hipMemcpyDeviceToHost);

	for (int i = 0; i < ref->_num_of_refs; i++) {
		hipMemcpy(ref->_nodes_data_ref[i], h_origins_refs[i], entry_size * ref->_ref_sizes[i], hipMemcpyDeviceToHost);
	}

	free(h_origins_refs);
}

void RainbowTable_cuda_free(rainbow_table_t* ref_rainbow_table, unsigned char** keys_pointers, unsigned char** encoded_passwords_pointers, unsigned char** origins_refs) {
	rainbow_table_t* ref = ref_rainbow_table;

	hipFree(keys_pointers);
	hipFree(encoded_passwords_pointers);

	unsigned char** h_origins_refs = (unsigned char**)malloc(sizeof(unsigned char*) * ref->_num_of_refs);
	hipMemcpy(h_origins_refs, origins_refs, sizeof(unsigned char*) * ref->_num_of_refs, hipMemcpyDeviceToHost);

	for (int i = 0; i < ref->_num_of_refs; i++) {
		hipFree(h_origins_refs[i]);
	}
	hipFree(origins_refs);
}