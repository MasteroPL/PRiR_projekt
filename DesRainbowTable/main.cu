﻿#include "hip/hip_runtime.h"
#include ""
#include "rainbow_table.h"

#include <stdio.h>
#include <stdlib.h>
// do testów time
#include <time.h>
#include <hip/hip_runtime.h>

#include "indexer.cuh"
#include "kernel.h"

int main() {

	rainbow_table_t* t = RainbowTable_allocate(8, 16, 1000000);

	for (int i = 0; i < 20; i++) {

		t->nodes[i].key[0] = (char)(65 + i);
		t->nodes[i].encoded_password[0] = (char)(97 + i);
		t->nodes[i].key[7] = '\0';
		t->nodes[i].encoded_password[7] = '\0';

		for (int j = 0; j < 6; j++) {
			t->nodes[i].key[1 + j] = (char)(48 + j);
			t->nodes[i].encoded_password[1 + j] = (char)(48 + j);
		}
	}



	// testowanie dla jednego watku
	srand(time(NULL));
	unsigned char key_host[64];
	for (int i = 0; i < 64; i++) {
		key_host[i] = rand() % 2;
	}
	unsigned char plain_host[] = { 'm','e','n','d','a','1','2','3' };
	
	unsigned char* encrypted_host = (unsigned char*)malloc(sizeof(unsigned char)*8);

	//alokowanie do GPU
	unsigned char* key;
	hipMalloc(&key, sizeof(unsigned char) * 64);
	unsigned char* plain;
	hipMalloc(&plain, sizeof(unsigned char) * 8);
	unsigned char* encrypted;
	hipMalloc(&encrypted, sizeof(unsigned char) * 8);

	hipMemcpy(key, key_host, sizeof(unsigned char) * 64, hipMemcpyHostToDevice);
	hipMemcpy(plain, plain_host, sizeof(unsigned char) * 8, hipMemcpyHostToDevice);

	printf("%s\n", plain);
	hipMemcpy(encrypted_host, encrypted, sizeof(unsigned char) * 8, hipMemcpyDeviceToHost);


	DESCipher<<<1,1>>>(key,plain,encrypted);
	hipFree(key);
	hipFree(plain);
	hipFree(encrypted);
	printf("%s\n", encrypted_host);

	free(plain_host);
	free(key_host);
	free(encrypted_host);




	for (int i = 0; i < 20; i++) {
		printf("%i: %s | %s\n", i, t->nodes[i].key, t->nodes[i].encoded_password);
	}



	RainbowTable_free(t);

	return 0;
}